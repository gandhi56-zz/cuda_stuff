#include <hip/hip_runtime.h>
#include "DeviceQuery.h"
#include <stdio.h>

int SimpleDeviceQuery::run(){
    int iDev = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, iDev);

    printf("Device %d: %s\n", iDev, iProp.name);
    printf("Number of multiprocessors: %d\n", iProp.multiProcessorCount);
    printf("Total amount of Constant Memory: %4.2f KB\n", iProp.totalConstMem / 1024.0);
    printf("Total amount of Shared memory per block: %4.2f KB\n", iProp.sharedMemPerBlock/1024.0);
    printf("Total number of registers per block: %d\n", iProp.regsPerBlock);
    printf("Warp size: %d\n", iProp.warpSize);
    printf("Maximum number of threads per block: %d\n", iProp.maxThreadsPerBlock);
    printf("Maximum number of threads per multiprocessor: %d\n", iProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of warps per multiprocessor: %d\n", iProp.maxThreadsPerMultiProcessor / 32);
    return 0;
}